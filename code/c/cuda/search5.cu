#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <string>
#include <stdlib.h>
#include <time.h>
using namespace std;
#include "target.h"
#include "cpucycles.h"


class uint32 {
  unsigned int x;
public:
  unsigned int uint() { return x; }
  inline uint32() { }
  inline uint32(unsigned int u) { x = u; }
  inline uint32(const uint32 &a) { x = a.x; }
  friend inline uint32 operator+(uint32 a,uint32 b) { return a.x + b.x; }
  friend inline uint32 operator|(uint32 a,uint32 b) { return a.x | b.x; }
  friend inline uint32 operator&(uint32 a,uint32 b) { return a.x & b.x; }
  friend inline uint32 operator^(uint32 a,uint32 b) { return a.x ^ b.x; }
  friend inline uint32 andnot(uint32 a,uint32 b) { return a.x & ~b.x; }
  friend inline uint32 rotate1(uint32 a) { return (a.x << 1) | (a.x >> 31); }
  friend inline uint32 rotate5(uint32 a) { return (a.x << 5) | (a.x >> 27); }
  friend inline uint32 rotate30(uint32 a) { return (a.x << 30) | (a.x >> 2); }
  friend ostream& operator<<(ostream& o,const uint32& u) {
    o << hex << setw(2) << setfill('0') << ((u.x >> 24) & 255);
    o << hex << setw(2) << setfill('0') << ((u.x >> 16) & 255);
    o << hex << setw(2) << setfill('0') << ((u.x >> 8) & 255);
    o << hex << setw(2) << setfill('0') << ((u.x) & 255);
    return o;
  }
} ;

class hash {
  uint32 state[5];
public:
  hash() { }
  hash(const hash &x) {
    state[0] = x.state[0];
    state[1] = x.state[1];
    state[2] = x.state[2];
    state[3] = x.state[3];
    state[4] = x.state[4];
  }
  hash(const unsigned int x[5]) {
    state[0] = x[0];
    state[1] = x[1];
    state[2] = x[2];
    state[3] = x[3];
    state[4] = x[4];
  }
  void init()
  {
    state[0] = 0x67452301;
    state[1] = 0xefcdab89;
    state[2] = 0x98badcfe;
    state[3] = 0x10325476;
    state[4] = 0xc3d2e1f0;
  }
  int hammingdistance(hash b) {
    unsigned int x0 = (state[0] ^ b.state[0]).uint();
    unsigned int x1 = (state[1] ^ b.state[1]).uint();
    unsigned int x2 = (state[2] ^ b.state[2]).uint();
    unsigned int x3 = (state[3] ^ b.state[3]).uint();
    unsigned int x4 = (state[4] ^ b.state[4]).uint();
    // 32 1-bit chunks
    x0 = (x0 & 0x55555555) + ((x0 >> 1) & 0x55555555);
    x1 = (x1 & 0x55555555) + ((x1 >> 1) & 0x55555555);
    x2 = (x2 & 0x55555555) + ((x2 >> 1) & 0x55555555);
    x3 = (x3 & 0x55555555) + ((x3 >> 1) & 0x55555555);
    x4 = (x4 & 0x55555555) + ((x4 >> 1) & 0x55555555);
    // 16 2-bit chunks: 012,012,012,012,012,012,012,012,012,012,012,012,012,012,012,012
    x0 = (x0 & 0x33333333) + ((x0 >> 2) & 0x33333333);
    x1 = (x1 & 0x33333333) + ((x1 >> 2) & 0x33333333);
    x2 = (x2 & 0x33333333) + ((x2 >> 2) & 0x33333333);
    x3 = (x3 & 0x33333333) + ((x3 >> 2) & 0x33333333);
    x4 = (x4 & 0x33333333) + ((x4 >> 2) & 0x33333333);
    // 8 4-bit chunks: 01234,01234,01234,01234,01234,01234,01234,01234
    x0 = (x0 & 0x0f0f0f0f) + ((x0 >> 4) & 0x0f0f0f0f);
    x1 = (x1 & 0x0f0f0f0f) + ((x1 >> 4) & 0x0f0f0f0f);
    x2 = (x2 & 0x0f0f0f0f) + ((x2 >> 4) & 0x0f0f0f0f);
    x3 = (x3 & 0x0f0f0f0f) + ((x3 >> 4) & 0x0f0f0f0f);
    x4 = (x4 & 0x0f0f0f0f) + ((x4 >> 4) & 0x0f0f0f0f);
    // 4 8-bit chunks: 012345678,012345678,012345678,012345678
    x0 = (x0 * 16843009) >> 24;
    x1 = (x1 * 16843009) >> 24;
    x2 = (x2 * 16843009) >> 24;
    x3 = (x3 * 16843009) >> 24;
    x4 = (x4 * 16843009) >> 24;
    return x0 + x1 + x2 + x3 + x4;
  }
  friend ostream& operator<<(ostream& o,const hash& h) {
    o << h.state[0];
    o << h.state[1];
    o << h.state[2];
    o << h.state[3];
    o << h.state[4];
    return o;
  }
  void update(const unsigned int *in,unsigned long long inblocks) {
    uint32 a = state[0];
    uint32 b = state[1];
    uint32 c = state[2];
    uint32 d = state[3];
    uint32 e = state[4];
    uint32 f;
    uint32 x0,x1,x2,x3,x4,x5,x6,x7,x8,x9,x10,x11,x12,x13,x14,x15;
  
    while (inblocks > 0) {
      x0 = in[0];
      f = (c & b) | andnot(d,b);
      e = rotate5(a) + f + e + 0x5a827999 + x0;
      b = rotate30(b);
      x1 = in[1];
      f = (b & a) | andnot(c,a);
      d = rotate5(e) + f + d + 0x5a827999 + x1;
      a = rotate30(a);
      x2 = in[2];
      f = (a & e) | andnot(b,e);
      c = rotate5(d) + f + c + 0x5a827999 + x2;
      e = rotate30(e);
      x3 = in[3];
      f = (e & d) | andnot(a,d);
      b = rotate5(c) + f + b + 0x5a827999 + x3;
      d = rotate30(d);
      x4 = in[4];
      f = (d & c) | andnot(e,c);
      a = rotate5(b) + f + a + 0x5a827999 + x4;
      c = rotate30(c);
      x5 = in[5];
      f = (c & b) | andnot(d,b);
      e = rotate5(a) + f + e + 0x5a827999 + x5;
      b = rotate30(b);
      x6 = in[6];
      f = (b & a) | andnot(c,a);
      d = rotate5(e) + f + d + 0x5a827999 + x6;
      a = rotate30(a);
      x7 = in[7];
      f = (a & e) | andnot(b,e);
      c = rotate5(d) + f + c + 0x5a827999 + x7;
      e = rotate30(e);
      x8 = in[8];
      f = (e & d) | andnot(a,d);
      b = rotate5(c) + f + b + 0x5a827999 + x8;
      d = rotate30(d);
      x9 = in[9];
      f = (d & c) | andnot(e,c);
      a = rotate5(b) + f + a + 0x5a827999 + x9;
      c = rotate30(c);
      x10 = in[10];
      f = (c & b) | andnot(d,b);
      e = rotate5(a) + f + e + 0x5a827999 + x10;
      b = rotate30(b);
      x11 = in[11];
      f = (b & a) | andnot(c,a);
      d = rotate5(e) + f + d + 0x5a827999 + x11;
      a = rotate30(a);
      x12 = in[12];
      f = (a & e) | andnot(b,e);
      c = rotate5(d) + f + c + 0x5a827999 + x12;
      e = rotate30(e);
      x13 = in[13];
      f = (e & d) | andnot(a,d);
      b = rotate5(c) + f + b + 0x5a827999 + x13;
      d = rotate30(d);
      x14 = in[14];
      f = (d & c) | andnot(e,c);
      a = rotate5(b) + f + a + 0x5a827999 + x14;
      c = rotate30(c);
      x15 = in[15];
      f = (c & b) | andnot(d,b);
      e = rotate5(a) + f + e + 0x5a827999 + x15;
      b = rotate30(b);
      x0 = rotate1(x13 ^ x8 ^ x2 ^ x0);
      f = (b & a) | andnot(c,a);
      d = rotate5(e) + f + d + 0x5a827999 + x0;
      a = rotate30(a);
      x1 = rotate1(x14 ^ x9 ^ x3 ^ x1);
      f = (a & e) | andnot(b,e);
      c = rotate5(d) + f + c + 0x5a827999 + x1;
      e = rotate30(e);
      x2 = rotate1(x15 ^ x10 ^ x4 ^ x2);
      f = (e & d) | andnot(a,d);
      b = rotate5(c) + f + b + 0x5a827999 + x2;
      d = rotate30(d);
      x3 = rotate1(x0 ^ x11 ^ x5 ^ x3);
      f = (d & c) | andnot(e,c);
      a = rotate5(b) + f + a + 0x5a827999 + x3;
      c = rotate30(c);
      x4 = rotate1(x1 ^ x12 ^ x6 ^ x4);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0x6ed9eba1 + x4;
      b = rotate30(b);
      x5 = rotate1(x2 ^ x13 ^ x7 ^ x5);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0x6ed9eba1 + x5;
      a = rotate30(a);
      x6 = rotate1(x3 ^ x14 ^ x8 ^ x6);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0x6ed9eba1 + x6;
      e = rotate30(e);
      x7 = rotate1(x4 ^ x15 ^ x9 ^ x7);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0x6ed9eba1 + x7;
      d = rotate30(d);
      x8 = rotate1(x5 ^ x0 ^ x10 ^ x8);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0x6ed9eba1 + x8;
      c = rotate30(c);
      x9 = rotate1(x6 ^ x1 ^ x11 ^ x9);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0x6ed9eba1 + x9;
      b = rotate30(b);
      x10 = rotate1(x7 ^ x2 ^ x12 ^ x10);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0x6ed9eba1 + x10;
      a = rotate30(a);
      x11 = rotate1(x8 ^ x3 ^ x13 ^ x11);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0x6ed9eba1 + x11;
      e = rotate30(e);
      x12 = rotate1(x9 ^ x4 ^ x14 ^ x12);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0x6ed9eba1 + x12;
      d = rotate30(d);
      x13 = rotate1(x10 ^ x5 ^ x15 ^ x13);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0x6ed9eba1 + x13;
      c = rotate30(c);
      x14 = rotate1(x11 ^ x6 ^ x0 ^ x14);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0x6ed9eba1 + x14;
      b = rotate30(b);
      x15 = rotate1(x12 ^ x7 ^ x1 ^ x15);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0x6ed9eba1 + x15;
      a = rotate30(a);
      x0 = rotate1(x13 ^ x8 ^ x2 ^ x0);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0x6ed9eba1 + x0;
      e = rotate30(e);
      x1 = rotate1(x14 ^ x9 ^ x3 ^ x1);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0x6ed9eba1 + x1;
      d = rotate30(d);
      x2 = rotate1(x15 ^ x10 ^ x4 ^ x2);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0x6ed9eba1 + x2;
      c = rotate30(c);
      x3 = rotate1(x0 ^ x11 ^ x5 ^ x3);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0x6ed9eba1 + x3;
      b = rotate30(b);
      x4 = rotate1(x1 ^ x12 ^ x6 ^ x4);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0x6ed9eba1 + x4;
      a = rotate30(a);
      x5 = rotate1(x2 ^ x13 ^ x7 ^ x5);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0x6ed9eba1 + x5;
      e = rotate30(e);
      x6 = rotate1(x3 ^ x14 ^ x8 ^ x6);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0x6ed9eba1 + x6;
      d = rotate30(d);
      x7 = rotate1(x4 ^ x15 ^ x9 ^ x7);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0x6ed9eba1 + x7;
      c = rotate30(c);
      x8 = rotate1(x5 ^ x0 ^ x10 ^ x8);
      f = (b & c) | (b & d) | (c & d);
      e = rotate5(a) + f + e + 0x8f1bbcdc + x8;
      b = rotate30(b);
      x9 = rotate1(x6 ^ x1 ^ x11 ^ x9);
      f = (a & b) | (a & c) | (b & c);
      d = rotate5(e) + f + d + 0x8f1bbcdc + x9;
      a = rotate30(a);
      x10 = rotate1(x7 ^ x2 ^ x12 ^ x10);
      f = (e & a) | (e & b) | (a & b);
      c = rotate5(d) + f + c + 0x8f1bbcdc + x10;
      e = rotate30(e);
      x11 = rotate1(x8 ^ x3 ^ x13 ^ x11);
      f = (d & e) | (d & a) | (e & a);
      b = rotate5(c) + f + b + 0x8f1bbcdc + x11;
      d = rotate30(d);
      x12 = rotate1(x9 ^ x4 ^ x14 ^ x12);
      f = (c & d) | (c & e) | (d & e);
      a = rotate5(b) + f + a + 0x8f1bbcdc + x12;
      c = rotate30(c);
      x13 = rotate1(x10 ^ x5 ^ x15 ^ x13);
      f = (b & c) | (b & d) | (c & d);
      e = rotate5(a) + f + e + 0x8f1bbcdc + x13;
      b = rotate30(b);
      x14 = rotate1(x11 ^ x6 ^ x0 ^ x14);
      f = (a & b) | (a & c) | (b & c);
      d = rotate5(e) + f + d + 0x8f1bbcdc + x14;
      a = rotate30(a);
      x15 = rotate1(x12 ^ x7 ^ x1 ^ x15);
      f = (e & a) | (e & b) | (a & b);
      c = rotate5(d) + f + c + 0x8f1bbcdc + x15;
      e = rotate30(e);
      x0 = rotate1(x13 ^ x8 ^ x2 ^ x0);
      f = (d & e) | (d & a) | (e & a);
      b = rotate5(c) + f + b + 0x8f1bbcdc + x0;
      d = rotate30(d);
      x1 = rotate1(x14 ^ x9 ^ x3 ^ x1);
      f = (c & d) | (c & e) | (d & e);
      a = rotate5(b) + f + a + 0x8f1bbcdc + x1;
      c = rotate30(c);
      x2 = rotate1(x15 ^ x10 ^ x4 ^ x2);
      f = (b & c) | (b & d) | (c & d);
      e = rotate5(a) + f + e + 0x8f1bbcdc + x2;
      b = rotate30(b);
      x3 = rotate1(x0 ^ x11 ^ x5 ^ x3);
      f = (a & b) | (a & c) | (b & c);
      d = rotate5(e) + f + d + 0x8f1bbcdc + x3;
      a = rotate30(a);
      x4 = rotate1(x1 ^ x12 ^ x6 ^ x4);
      f = (e & a) | (e & b) | (a & b);
      c = rotate5(d) + f + c + 0x8f1bbcdc + x4;
      e = rotate30(e);
      x5 = rotate1(x2 ^ x13 ^ x7 ^ x5);
      f = (d & e) | (d & a) | (e & a);
      b = rotate5(c) + f + b + 0x8f1bbcdc + x5;
      d = rotate30(d);
      x6 = rotate1(x3 ^ x14 ^ x8 ^ x6);
      f = (c & d) | (c & e) | (d & e);
      a = rotate5(b) + f + a + 0x8f1bbcdc + x6;
      c = rotate30(c);
      x7 = rotate1(x4 ^ x15 ^ x9 ^ x7);
      f = (b & c) | (b & d) | (c & d);
      e = rotate5(a) + f + e + 0x8f1bbcdc + x7;
      b = rotate30(b);
      x8 = rotate1(x5 ^ x0 ^ x10 ^ x8);
      f = (a & b) | (a & c) | (b & c);
      d = rotate5(e) + f + d + 0x8f1bbcdc + x8;
      a = rotate30(a);
      x9 = rotate1(x6 ^ x1 ^ x11 ^ x9);
      f = (e & a) | (e & b) | (a & b);
      c = rotate5(d) + f + c + 0x8f1bbcdc + x9;
      e = rotate30(e);
      x10 = rotate1(x7 ^ x2 ^ x12 ^ x10);
      f = (d & e) | (d & a) | (e & a);
      b = rotate5(c) + f + b + 0x8f1bbcdc + x10;
      d = rotate30(d);
      x11 = rotate1(x8 ^ x3 ^ x13 ^ x11);
      f = (c & d) | (c & e) | (d & e);
      a = rotate5(b) + f + a + 0x8f1bbcdc + x11;
      c = rotate30(c);
      x12 = rotate1(x9 ^ x4 ^ x14 ^ x12);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0xca62c1d6 + x12;
      b = rotate30(b);
      x13 = rotate1(x10 ^ x5 ^ x15 ^ x13);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0xca62c1d6 + x13;
      a = rotate30(a);
      x14 = rotate1(x11 ^ x6 ^ x0 ^ x14);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0xca62c1d6 + x14;
      e = rotate30(e);
      x15 = rotate1(x12 ^ x7 ^ x1 ^ x15);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0xca62c1d6 + x15;
      d = rotate30(d);
      x0 = rotate1(x13 ^ x8 ^ x2 ^ x0);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0xca62c1d6 + x0;
      c = rotate30(c);
      x1 = rotate1(x14 ^ x9 ^ x3 ^ x1);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0xca62c1d6 + x1;
      b = rotate30(b);
      x2 = rotate1(x15 ^ x10 ^ x4 ^ x2);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0xca62c1d6 + x2;
      a = rotate30(a);
      x3 = rotate1(x0 ^ x11 ^ x5 ^ x3);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0xca62c1d6 + x3;
      e = rotate30(e);
      x4 = rotate1(x1 ^ x12 ^ x6 ^ x4);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0xca62c1d6 + x4;
      d = rotate30(d);
      x5 = rotate1(x2 ^ x13 ^ x7 ^ x5);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0xca62c1d6 + x5;
      c = rotate30(c);
      x6 = rotate1(x3 ^ x14 ^ x8 ^ x6);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0xca62c1d6 + x6;
      b = rotate30(b);
      x7 = rotate1(x4 ^ x15 ^ x9 ^ x7);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0xca62c1d6 + x7;
      a = rotate30(a);
      x8 = rotate1(x5 ^ x0 ^ x10 ^ x8);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0xca62c1d6 + x8;
      e = rotate30(e);
      x9 = rotate1(x6 ^ x1 ^ x11 ^ x9);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0xca62c1d6 + x9;
      d = rotate30(d);
      x10 = rotate1(x7 ^ x2 ^ x12 ^ x10);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0xca62c1d6 + x10;
      c = rotate30(c);
      x11 = rotate1(x8 ^ x3 ^ x13 ^ x11);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0xca62c1d6 + x11;
      b = rotate30(b);
      x12 = rotate1(x9 ^ x4 ^ x14 ^ x12);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0xca62c1d6 + x12;
      a = rotate30(a);
      x13 = rotate1(x10 ^ x5 ^ x15 ^ x13);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0xca62c1d6 + x13;
      e = rotate30(e);
      x14 = rotate1(x11 ^ x6 ^ x0 ^ x14);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0xca62c1d6 + x14;
      d = rotate30(d);
      x15 = rotate1(x12 ^ x7 ^ x1 ^ x15);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0xca62c1d6 + x15;
      c = rotate30(c);
  
      a = a + state[0];
      b = b + state[1];
      c = c + state[2];
      d = d + state[3];
      e = e + state[4];
      state[0] = a;
      state[1] = b;
      state[2] = c;
      state[3] = d;
      state[4] = e;
  
      --inblocks;
      in += 16; 
    }
  }
} ;

__device__ unsigned int andnot(unsigned int a,unsigned int b)
{
  return a & ~b;
}

__device__ unsigned int rotate1(unsigned int a)
{
  return (a << 1) | (a >> 31);
}

__device__ unsigned int rotate5(unsigned int a)
{
  return (a << 5) | (a >> 27);
}

__device__ unsigned int rotate30(unsigned int a)
{
  return (a << 30) | (a >> 2);
}

class gpu_hash {
  unsigned int state0;
  unsigned int state1;
  unsigned int state2;
  unsigned int state3;
  unsigned int state4;
public:
  __device__ gpu_hash() { }
  __device__ gpu_hash(const gpu_hash &x) {
    state0 = x.state0;
    state1 = x.state1;
    state2 = x.state2;
    state3 = x.state3;
    state4 = x.state4;
  }
  __device__ gpu_hash(const unsigned int x[5]) {
    state0 = x[0];
    state1 = x[1];
    state2 = x[2];
    state3 = x[3];
    state4 = x[4];
  }
  __device__ void init()
  {
    state0 = 0x67452301;
    state1 = 0xefcdab89;
    state2 = 0x98badcfe;
    state3 = 0x10325476;
    state4 = 0xc3d2e1f0;
  }
  __device__ int hammingdistance(gpu_hash b) {
    unsigned int x0 = (state0 ^ b.state0);
    unsigned int x1 = (state1 ^ b.state1);
    unsigned int x2 = (state2 ^ b.state2);
    unsigned int x3 = (state3 ^ b.state3);
    unsigned int x4 = (state4 ^ b.state4);
    // 32 1-bit chunks
    x0 = (x0 & 0x55555555) + ((x0 >> 1) & 0x55555555);
    x1 = (x1 & 0x55555555) + ((x1 >> 1) & 0x55555555);
    x2 = (x2 & 0x55555555) + ((x2 >> 1) & 0x55555555);
    x3 = (x3 & 0x55555555) + ((x3 >> 1) & 0x55555555);
    x4 = (x4 & 0x55555555) + ((x4 >> 1) & 0x55555555);
    // 16 2-bit chunks: 012,012,012,012,012,012,012,012,012,012,012,012,012,012,012,012
    x0 = (x0 & 0x33333333) + ((x0 >> 2) & 0x33333333);
    x1 = (x1 & 0x33333333) + ((x1 >> 2) & 0x33333333);
    x2 = (x2 & 0x33333333) + ((x2 >> 2) & 0x33333333);
    x3 = (x3 & 0x33333333) + ((x3 >> 2) & 0x33333333);
    x4 = (x4 & 0x33333333) + ((x4 >> 2) & 0x33333333);
    // 8 4-bit chunks: 01234,01234,01234,01234,01234,01234,01234,01234
    x0 = (x0 & 0x0f0f0f0f) + ((x0 >> 4) & 0x0f0f0f0f);
    x1 = (x1 & 0x0f0f0f0f) + ((x1 >> 4) & 0x0f0f0f0f);
    x2 = (x2 & 0x0f0f0f0f) + ((x2 >> 4) & 0x0f0f0f0f);
    x3 = (x3 & 0x0f0f0f0f) + ((x3 >> 4) & 0x0f0f0f0f);
    x4 = (x4 & 0x0f0f0f0f) + ((x4 >> 4) & 0x0f0f0f0f);
    // 4 8-bit chunks: 012345678,012345678,012345678,012345678
    x0 = (x0 * 16843009) >> 24;
    x1 = (x1 * 16843009) >> 24;
    x2 = (x2 * 16843009) >> 24;
    x3 = (x3 * 16843009) >> 24;
    x4 = (x4 * 16843009) >> 24;
    return x0 + x1 + x2 + x3 + x4;
  }
  __device__ void update(const unsigned int *in,unsigned long long inblocks) {
    unsigned int a = state0;
    unsigned int b = state1;
    unsigned int c = state2;
    unsigned int d = state3;
    unsigned int e = state4;
    unsigned int f;
    unsigned int x0,x1,x2,x3,x4,x5,x6,x7,x8,x9,x10,x11,x12,x13,x14,x15;
  
    while (inblocks > 0) {
      x0 = in[0];
      f = (c & b) | andnot(d,b);
      e = rotate5(a) + f + e + 0x5a827999 + x0;
      b = rotate30(b);
      x1 = in[1];
      f = (b & a) | andnot(c,a);
      d = rotate5(e) + f + d + 0x5a827999 + x1;
      a = rotate30(a);
      x2 = in[2];
      f = (a & e) | andnot(b,e);
      c = rotate5(d) + f + c + 0x5a827999 + x2;
      e = rotate30(e);
      x3 = in[3];
      f = (e & d) | andnot(a,d);
      b = rotate5(c) + f + b + 0x5a827999 + x3;
      d = rotate30(d);
      x4 = in[4];
      f = (d & c) | andnot(e,c);
      a = rotate5(b) + f + a + 0x5a827999 + x4;
      c = rotate30(c);
      x5 = in[5];
      f = (c & b) | andnot(d,b);
      e = rotate5(a) + f + e + 0x5a827999 + x5;
      b = rotate30(b);
      x6 = in[6];
      f = (b & a) | andnot(c,a);
      d = rotate5(e) + f + d + 0x5a827999 + x6;
      a = rotate30(a);
      x7 = in[7];
      f = (a & e) | andnot(b,e);
      c = rotate5(d) + f + c + 0x5a827999 + x7;
      e = rotate30(e);
      x8 = in[8];
      f = (e & d) | andnot(a,d);
      b = rotate5(c) + f + b + 0x5a827999 + x8;
      d = rotate30(d);
      x9 = in[9];
      f = (d & c) | andnot(e,c);
      a = rotate5(b) + f + a + 0x5a827999 + x9;
      c = rotate30(c);
      x10 = in[10];
      f = (c & b) | andnot(d,b);
      e = rotate5(a) + f + e + 0x5a827999 + x10;
      b = rotate30(b);
      x11 = in[11];
      f = (b & a) | andnot(c,a);
      d = rotate5(e) + f + d + 0x5a827999 + x11;
      a = rotate30(a);
      x12 = in[12];
      f = (a & e) | andnot(b,e);
      c = rotate5(d) + f + c + 0x5a827999 + x12;
      e = rotate30(e);
      x13 = in[13];
      f = (e & d) | andnot(a,d);
      b = rotate5(c) + f + b + 0x5a827999 + x13;
      d = rotate30(d);
      x14 = in[14];
      f = (d & c) | andnot(e,c);
      a = rotate5(b) + f + a + 0x5a827999 + x14;
      c = rotate30(c);
      x15 = in[15];
      f = (c & b) | andnot(d,b);
      e = rotate5(a) + f + e + 0x5a827999 + x15;
      b = rotate30(b);
      x0 = rotate1(x13 ^ x8 ^ x2 ^ x0);
      f = (b & a) | andnot(c,a);
      d = rotate5(e) + f + d + 0x5a827999 + x0;
      a = rotate30(a);
      x1 = rotate1(x14 ^ x9 ^ x3 ^ x1);
      f = (a & e) | andnot(b,e);
      c = rotate5(d) + f + c + 0x5a827999 + x1;
      e = rotate30(e);
      x2 = rotate1(x15 ^ x10 ^ x4 ^ x2);
      f = (e & d) | andnot(a,d);
      b = rotate5(c) + f + b + 0x5a827999 + x2;
      d = rotate30(d);
      x3 = rotate1(x0 ^ x11 ^ x5 ^ x3);
      f = (d & c) | andnot(e,c);
      a = rotate5(b) + f + a + 0x5a827999 + x3;
      c = rotate30(c);
      x4 = rotate1(x1 ^ x12 ^ x6 ^ x4);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0x6ed9eba1 + x4;
      b = rotate30(b);
      x5 = rotate1(x2 ^ x13 ^ x7 ^ x5);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0x6ed9eba1 + x5;
      a = rotate30(a);
      x6 = rotate1(x3 ^ x14 ^ x8 ^ x6);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0x6ed9eba1 + x6;
      e = rotate30(e);
      x7 = rotate1(x4 ^ x15 ^ x9 ^ x7);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0x6ed9eba1 + x7;
      d = rotate30(d);
      x8 = rotate1(x5 ^ x0 ^ x10 ^ x8);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0x6ed9eba1 + x8;
      c = rotate30(c);
      x9 = rotate1(x6 ^ x1 ^ x11 ^ x9);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0x6ed9eba1 + x9;
      b = rotate30(b);
      x10 = rotate1(x7 ^ x2 ^ x12 ^ x10);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0x6ed9eba1 + x10;
      a = rotate30(a);
      x11 = rotate1(x8 ^ x3 ^ x13 ^ x11);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0x6ed9eba1 + x11;
      e = rotate30(e);
      x12 = rotate1(x9 ^ x4 ^ x14 ^ x12);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0x6ed9eba1 + x12;
      d = rotate30(d);
      x13 = rotate1(x10 ^ x5 ^ x15 ^ x13);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0x6ed9eba1 + x13;
      c = rotate30(c);
      x14 = rotate1(x11 ^ x6 ^ x0 ^ x14);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0x6ed9eba1 + x14;
      b = rotate30(b);
      x15 = rotate1(x12 ^ x7 ^ x1 ^ x15);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0x6ed9eba1 + x15;
      a = rotate30(a);
      x0 = rotate1(x13 ^ x8 ^ x2 ^ x0);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0x6ed9eba1 + x0;
      e = rotate30(e);
      x1 = rotate1(x14 ^ x9 ^ x3 ^ x1);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0x6ed9eba1 + x1;
      d = rotate30(d);
      x2 = rotate1(x15 ^ x10 ^ x4 ^ x2);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0x6ed9eba1 + x2;
      c = rotate30(c);
      x3 = rotate1(x0 ^ x11 ^ x5 ^ x3);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0x6ed9eba1 + x3;
      b = rotate30(b);
      x4 = rotate1(x1 ^ x12 ^ x6 ^ x4);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0x6ed9eba1 + x4;
      a = rotate30(a);
      x5 = rotate1(x2 ^ x13 ^ x7 ^ x5);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0x6ed9eba1 + x5;
      e = rotate30(e);
      x6 = rotate1(x3 ^ x14 ^ x8 ^ x6);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0x6ed9eba1 + x6;
      d = rotate30(d);
      x7 = rotate1(x4 ^ x15 ^ x9 ^ x7);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0x6ed9eba1 + x7;
      c = rotate30(c);
      x8 = rotate1(x5 ^ x0 ^ x10 ^ x8);
      f = (b & c) | (b & d) | (c & d);
      e = rotate5(a) + f + e + 0x8f1bbcdc + x8;
      b = rotate30(b);
      x9 = rotate1(x6 ^ x1 ^ x11 ^ x9);
      f = (a & b) | (a & c) | (b & c);
      d = rotate5(e) + f + d + 0x8f1bbcdc + x9;
      a = rotate30(a);
      x10 = rotate1(x7 ^ x2 ^ x12 ^ x10);
      f = (e & a) | (e & b) | (a & b);
      c = rotate5(d) + f + c + 0x8f1bbcdc + x10;
      e = rotate30(e);
      x11 = rotate1(x8 ^ x3 ^ x13 ^ x11);
      f = (d & e) | (d & a) | (e & a);
      b = rotate5(c) + f + b + 0x8f1bbcdc + x11;
      d = rotate30(d);
      x12 = rotate1(x9 ^ x4 ^ x14 ^ x12);
      f = (c & d) | (c & e) | (d & e);
      a = rotate5(b) + f + a + 0x8f1bbcdc + x12;
      c = rotate30(c);
      x13 = rotate1(x10 ^ x5 ^ x15 ^ x13);
      f = (b & c) | (b & d) | (c & d);
      e = rotate5(a) + f + e + 0x8f1bbcdc + x13;
      b = rotate30(b);
      x14 = rotate1(x11 ^ x6 ^ x0 ^ x14);
      f = (a & b) | (a & c) | (b & c);
      d = rotate5(e) + f + d + 0x8f1bbcdc + x14;
      a = rotate30(a);
      x15 = rotate1(x12 ^ x7 ^ x1 ^ x15);
      f = (e & a) | (e & b) | (a & b);
      c = rotate5(d) + f + c + 0x8f1bbcdc + x15;
      e = rotate30(e);
      x0 = rotate1(x13 ^ x8 ^ x2 ^ x0);
      f = (d & e) | (d & a) | (e & a);
      b = rotate5(c) + f + b + 0x8f1bbcdc + x0;
      d = rotate30(d);
      x1 = rotate1(x14 ^ x9 ^ x3 ^ x1);
      f = (c & d) | (c & e) | (d & e);
      a = rotate5(b) + f + a + 0x8f1bbcdc + x1;
      c = rotate30(c);
      x2 = rotate1(x15 ^ x10 ^ x4 ^ x2);
      f = (b & c) | (b & d) | (c & d);
      e = rotate5(a) + f + e + 0x8f1bbcdc + x2;
      b = rotate30(b);
      x3 = rotate1(x0 ^ x11 ^ x5 ^ x3);
      f = (a & b) | (a & c) | (b & c);
      d = rotate5(e) + f + d + 0x8f1bbcdc + x3;
      a = rotate30(a);
      x4 = rotate1(x1 ^ x12 ^ x6 ^ x4);
      f = (e & a) | (e & b) | (a & b);
      c = rotate5(d) + f + c + 0x8f1bbcdc + x4;
      e = rotate30(e);
      x5 = rotate1(x2 ^ x13 ^ x7 ^ x5);
      f = (d & e) | (d & a) | (e & a);
      b = rotate5(c) + f + b + 0x8f1bbcdc + x5;
      d = rotate30(d);
      x6 = rotate1(x3 ^ x14 ^ x8 ^ x6);
      f = (c & d) | (c & e) | (d & e);
      a = rotate5(b) + f + a + 0x8f1bbcdc + x6;
      c = rotate30(c);
      x7 = rotate1(x4 ^ x15 ^ x9 ^ x7);
      f = (b & c) | (b & d) | (c & d);
      e = rotate5(a) + f + e + 0x8f1bbcdc + x7;
      b = rotate30(b);
      x8 = rotate1(x5 ^ x0 ^ x10 ^ x8);
      f = (a & b) | (a & c) | (b & c);
      d = rotate5(e) + f + d + 0x8f1bbcdc + x8;
      a = rotate30(a);
      x9 = rotate1(x6 ^ x1 ^ x11 ^ x9);
      f = (e & a) | (e & b) | (a & b);
      c = rotate5(d) + f + c + 0x8f1bbcdc + x9;
      e = rotate30(e);
      x10 = rotate1(x7 ^ x2 ^ x12 ^ x10);
      f = (d & e) | (d & a) | (e & a);
      b = rotate5(c) + f + b + 0x8f1bbcdc + x10;
      d = rotate30(d);
      x11 = rotate1(x8 ^ x3 ^ x13 ^ x11);
      f = (c & d) | (c & e) | (d & e);
      a = rotate5(b) + f + a + 0x8f1bbcdc + x11;
      c = rotate30(c);
      x12 = rotate1(x9 ^ x4 ^ x14 ^ x12);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0xca62c1d6 + x12;
      b = rotate30(b);
      x13 = rotate1(x10 ^ x5 ^ x15 ^ x13);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0xca62c1d6 + x13;
      a = rotate30(a);
      x14 = rotate1(x11 ^ x6 ^ x0 ^ x14);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0xca62c1d6 + x14;
      e = rotate30(e);
      x15 = rotate1(x12 ^ x7 ^ x1 ^ x15);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0xca62c1d6 + x15;
      d = rotate30(d);
      x0 = rotate1(x13 ^ x8 ^ x2 ^ x0);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0xca62c1d6 + x0;
      c = rotate30(c);
      x1 = rotate1(x14 ^ x9 ^ x3 ^ x1);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0xca62c1d6 + x1;
      b = rotate30(b);
      x2 = rotate1(x15 ^ x10 ^ x4 ^ x2);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0xca62c1d6 + x2;
      a = rotate30(a);
      x3 = rotate1(x0 ^ x11 ^ x5 ^ x3);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0xca62c1d6 + x3;
      e = rotate30(e);
      x4 = rotate1(x1 ^ x12 ^ x6 ^ x4);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0xca62c1d6 + x4;
      d = rotate30(d);
      x5 = rotate1(x2 ^ x13 ^ x7 ^ x5);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0xca62c1d6 + x5;
      c = rotate30(c);
      x6 = rotate1(x3 ^ x14 ^ x8 ^ x6);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0xca62c1d6 + x6;
      b = rotate30(b);
      x7 = rotate1(x4 ^ x15 ^ x9 ^ x7);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0xca62c1d6 + x7;
      a = rotate30(a);
      x8 = rotate1(x5 ^ x0 ^ x10 ^ x8);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0xca62c1d6 + x8;
      e = rotate30(e);
      x9 = rotate1(x6 ^ x1 ^ x11 ^ x9);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0xca62c1d6 + x9;
      d = rotate30(d);
      x10 = rotate1(x7 ^ x2 ^ x12 ^ x10);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0xca62c1d6 + x10;
      c = rotate30(c);
      x11 = rotate1(x8 ^ x3 ^ x13 ^ x11);
      f = b ^ c ^ d;
      e = rotate5(a) + f + e + 0xca62c1d6 + x11;
      b = rotate30(b);
      x12 = rotate1(x9 ^ x4 ^ x14 ^ x12);
      f = a ^ b ^ c;
      d = rotate5(e) + f + d + 0xca62c1d6 + x12;
      a = rotate30(a);
      x13 = rotate1(x10 ^ x5 ^ x15 ^ x13);
      f = e ^ a ^ b;
      c = rotate5(d) + f + c + 0xca62c1d6 + x13;
      e = rotate30(e);
      x14 = rotate1(x11 ^ x6 ^ x0 ^ x14);
      f = d ^ e ^ a;
      b = rotate5(c) + f + b + 0xca62c1d6 + x14;
      d = rotate30(d);
      x15 = rotate1(x12 ^ x7 ^ x1 ^ x15);
      f = c ^ d ^ e;
      a = rotate5(b) + f + a + 0xca62c1d6 + x15;
      c = rotate30(c);
  
      a = a + state0;
      b = b + state1;
      c = c + state2;
      d = d + state3;
      e = e + state4;
      state0 = a;
      state1 = b;
      state2 = c;
      state3 = d;
      state4 = e;
  
      --inblocks;
      in += 16; 
    }
  }
} ;

__constant__ const char ALPHABET[65] =
"abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789/_";

#define ALPHABETTOP 32
#define ALPHABETBOT 62

unsigned int words[64 + 16 * ((sizeof s + sizeof target) / 64)];

__global__ void doit(unsigned char *s,unsigned long long slen,
  unsigned int sblocks,unsigned int sblockspre,
  unsigned int *targetstate,unsigned int *results)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int c0;
  int c1;
  int c2;
  int c3;
  int c4;
  int i;
  unsigned int words[32];

  for (i = 0;i < sizeof words;++i) ((unsigned char *) words)[i] = 0;
  for (i = 0;i < slen - 5;++i) ((unsigned char *) words)[i ^ 3] = s[i];
  ((unsigned char *) words)[slen ^ 3] = 0x80;
  words[sblocks * 16 - 1] = slen * 8;

  gpu_hash shashpre;
  shashpre.init();
  shashpre.update(words,sblockspre);

  gpu_hash targethash(targetstate);

  c0 = tid % ALPHABETTOP;
  c1 = (tid / ALPHABETTOP) % ALPHABETTOP;
  c2 = ((tid / ALPHABETTOP) / ALPHABETTOP) % ALPHABETTOP;

  ((unsigned char *) words)[(slen - 5) ^ 3] = ALPHABET[c0];
  ((unsigned char *) words)[(slen - 4) ^ 3] = ALPHABET[c1];
  ((unsigned char *) words)[(slen - 3) ^ 3] = ALPHABET[c2];
  for (c3 = 0;c3 < ALPHABETBOT;++c3) {
    ((unsigned char *) words)[(slen - 2) ^ 3] = ALPHABET[c3];
    for (c4 = 0;c4 < ALPHABETBOT;++c4) {
      ((unsigned char *) words)[(slen - 1) ^ 3] = ALPHABET[c4];

      gpu_hash shash(shashpre);
      shash.update(words + sblockspre * 16,sblocks - sblockspre);
      int d = shash.hammingdistance(targethash);

      if (d < 38) {
        results[0] = tid;
        results[1] = ALPHABET[c0];
        results[2] = ALPHABET[c1];
        results[3] = ALPHABET[c2];
        results[4] = ALPHABET[c3];
        results[5] = ALPHABET[c4];
      }
    }
  }
}

int main(int argc,char **argv)
{
  int i;
  
  long long targetlen = 0;
  while (target[targetlen]) ++targetlen;
  long long targetblocks = (targetlen + 72) / 64;

  for (i = 0;i < sizeof words;++i) ((unsigned char *) words)[i] = 0;
  for (i = 0;i < targetlen;++i) ((unsigned char *) words)[i ^ 3] = target[i];
  ((unsigned char *) words)[targetlen ^ 3] = 0x80;
  words[targetblocks * 16 - 1] = targetlen * 8;

  hash targethash;
  targethash.init();
  targethash.update(words,targetblocks);

  cout << 0 << " " << targethash << " " << target << "\n";

  unsigned char flip[sizeof s];

  long long slen = 0;
  while (s[slen]) ++slen;
  if (slen < 5) return 100;
  long long sblocks = (slen + 72) / 64;
  long long sblockspre = (slen - 5) / 64;

#ifndef NONRANDOM
  srandom(cpucycles()); // XXX: randomize better
#endif
  for (i = 0;i < slen;++i) {
    flip[i] = 0;
    if (random() & 1) if (s[i] != ' ') s[i] ^= 32;
  }

  if (argv[1]) hipSetDevice(atoi(argv[1]));

  int numgpu;
  hipGetDeviceCount(&numgpu);
  int curgpu;
  hipGetDevice(&curgpu);
  cout << "using GPU " << curgpu << " out of " << numgpu << "\n";

  unsigned int *gpu_targethash;
  hipMalloc((void **) &gpu_targethash,5 * sizeof(unsigned int));

  unsigned int results[10];
  unsigned int *gpu_results;
  hipMalloc((void **) &gpu_results,10 * sizeof(unsigned int));

  unsigned char *gpu_s;
  hipMalloc((void **) &gpu_s,slen + 1);

  dim3 dimBlock(64);
  dim3 dimGrid((ALPHABETTOP * ALPHABETTOP * ALPHABETTOP) / 64);

  long long startcycles = cpucycles();
  long long innerloopcycles = 0;
  long long hashes = 1;
  long long printcycles = 0;

  for (;;) {
    ++printcycles;
    if (printcycles == 128) {
      cout << "cycles " << dec
        << (cpucycles() - startcycles) / (1.0 * hashes) << " "
        << (innerloopcycles) / (1.0 * hashes) << " "
        << s << "\n" << flush;
      printcycles = 0;
    }

    results[0] = 0;
    hipMemcpy(gpu_targethash,&targethash,sizeof targethash,hipMemcpyHostToDevice);
    hipMemcpy(gpu_s,s,slen + 1,hipMemcpyHostToDevice);
    hipMemcpy(gpu_results,results,sizeof results,hipMemcpyHostToDevice);
    innerloopcycles -= cpucycles();
    doit<<<dimGrid,dimBlock>>>(gpu_s,slen,sblocks,sblockspre,gpu_targethash,gpu_results);
    hipMemcpy(results,gpu_results,sizeof results,hipMemcpyDeviceToHost);
    innerloopcycles += cpucycles();

    hashes += ALPHABETTOP * (long long) ALPHABETTOP * (long long) ALPHABETTOP
      * (long long) ALPHABETBOT * (long long) ALPHABETBOT;

    if (results[0]) {
      s[slen - 5] = results[1];
      s[slen - 4] = results[2];
      s[slen - 3] = results[3];
      s[slen - 2] = results[4];
      s[slen - 1] = results[5];

      for (i = 0;i < sizeof words;++i) ((unsigned char *) words)[i] = 0;
      for (i = 0;i < slen;++i) ((unsigned char *) words)[i ^ 3] = s[i];
      ((unsigned char *) words)[slen ^ 3] = 0x80;
      words[sblocks * 16 - 1] = slen * 8;

      hash shash;
      shash.init();
      shash.update(words,sblocks);

      cout << dec << shash.hammingdistance(targethash) << " " << shash << " " << s << "\n" << flush;
    }

    for (i = 0;i < slen - 5;++i) if (s[i] != ' ') {
      s[i] ^= 32;
      flip[i] ^= 32;
      if (flip[i]) break;
    }

    if (i == slen - 5) return 0;
  }
}
